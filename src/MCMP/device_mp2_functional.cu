#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/device_allocator.h>
#include "hipblas.h"
#include "device_mp2_functional.h"

template <int CVMP2>
Device_MP2_Functional<CVMP2>::Device_MP2_Functional(int electron_pairs) : 
  Standard_MP_Functional<thrust::device_vector, thrust::device_allocator>(CVMP2 * (CVMP2+1), 1, "22"),
  vector_size(electron_pairs),
  matrix_size(vector_size * vector_size),
  ctrl(n_control_variates),
  o_direct(matrix_size),
  o_exchange(matrix_size),
  v_direct(matrix_size),
  v_exchange(matrix_size),
  scratch_matrix(matrix_size),
  scratch_vector(2 * vector_size),
  d_en_ctrl(8),
  h_en_ctrl(8)
{ 
  block_size = dim3(16, 16, 1);
  grid_size = dim3(
      (vector_size + block_size.x - 1) / block_size.x, 
      (vector_size + block_size.y - 1) / block_size.y, 
      1);
  hipblasCreate(&handle);
}

template <int CVMP2>
Device_MP2_Functional<CVMP2>::~Device_MP2_Functional() {
  hipblasDestroy(handle);
}

__global__ void m_m_add_mul(double alpha, double* A, double *B, double* C, int size) {
  int tidx = blockIdx.x * blockDim.x + threadIdx.x;
  int tidy = blockIdx.y * blockDim.y + threadIdx.y;
  int tid  = tidy * size + tidx;
  if(tidx < size && tidy < tidx) {
    C[tid] = alpha * A[tid] * B[tid] + C[tid];
  }
}

__global__ 
void mp2_functional_kernal(
    const double *o_direct,
    const double *o_exchange,
    const double *v_term,
    double *scratch_matrix,
    int size) {
  int tidx = blockIdx.x * blockDim.x + threadIdx.x;
  int tidy = blockIdx.y * blockDim.y + threadIdx.y;
  int tid  = tidy * size + tidx;
  if(tidx < size && tidy < tidx) {
    scratch_matrix[tid] -= 2.0 * o_direct[tid] * v_term[tid];
    scratch_matrix[tid] += o_exchange[tid] * v_term[tid];
  }
}

template <int CVMP2>
void Device_MP2_Functional<CVMP2>::prep_arrays(OVPS_Type& ovps, Electron_Pair_List_Type* electron_pair_list) {
  en2 = 0.0;
  std::fill(ctrl.begin(), ctrl.end(), 0.0);
  thrust::transform(ovps.o_set[0][0].s_11.begin(), ovps.o_set[0][0].s_11.end(), ovps.o_set[0][0].s_22.begin(), o_direct.begin(), thrust::multiplies<double>());
  thrust::transform(ovps.o_set[0][0].s_12.begin(), ovps.o_set[0][0].s_12.end(), ovps.o_set[0][0].s_21.begin(), o_exchange.begin(), thrust::multiplies<double>());
  thrust::transform(ovps.v_set[0][0].s_11.begin(), ovps.v_set[0][0].s_11.end(), ovps.v_set[0][0].s_22.begin(), v_direct.begin(), thrust::multiplies<double>());
  thrust::transform(ovps.v_set[0][0].s_12.begin(), ovps.v_set[0][0].s_12.end(), ovps.v_set[0][0].s_21.begin(), v_exchange.begin(), thrust::multiplies<double>());
}

template <int CVMP2>
void Device_MP2_Functional<CVMP2>::cv_energy_helper(int offset, const vector_double& rv_inverse_weight) {
  this->blas_wrapper.dgemm(false, false,
      vector_size, 2, vector_size,
      1.0,
      scratch_matrix, vector_size,
      rv_inverse_weight, vector_size,
      0.0,
      scratch_vector, vector_size);

  this->blas_wrapper.dgemm(true, false,
      2, 2, vector_size,
      1.0,
      scratch_vector, 0, vector_size,
      rv_inverse_weight, 0, vector_size,
      0.0,
      d_en_ctrl, offset * 2, 4);
}

template <int CVMP2>
void Device_MP2_Functional<CVMP2>::energy(double& emp, std::vector<double>& control, OVPS_Type& ovps, Electron_Pair_List_Type* electron_pair_list, Tau* tau) {
  prep_arrays(ovps, electron_pair_list);

  thrust::fill(scratch_matrix.begin(), scratch_matrix.end(), 0.0);
  m_m_add_mul<<<grid_size, block_size>>>(1.0, o_direct.data().get(), v_direct.data().get(), scratch_matrix.data().get(), vector_size);
  m_m_add_mul<<<grid_size, block_size>>>(1.0, o_exchange.data().get(), v_exchange.data().get(), scratch_matrix.data().get(), vector_size);
  cv_energy_helper(0, electron_pair_list->rv_inverse_weight);

  thrust::fill(scratch_matrix.begin(), scratch_matrix.end(), 0.0);
  m_m_add_mul<<<grid_size, block_size>>>(1.0, o_direct.data().get(),   v_exchange.data().get(), scratch_matrix.data().get(), vector_size);
  m_m_add_mul<<<grid_size, block_size>>>(1.0, o_exchange.data().get(), v_direct.data().get(), scratch_matrix.data().get(), vector_size);
  cv_energy_helper(1, electron_pair_list->rv_inverse_weight);

  thrust::copy(d_en_ctrl.begin(), d_en_ctrl.end(), h_en_ctrl.begin());
  en2 = h_en_ctrl[2] - 2.0 * h_en_ctrl[0];
  ctrl[0] = h_en_ctrl[1]; 
  ctrl[1] = h_en_ctrl[3];
  if (CVMP2 >= 2) {
    ctrl[2] = h_en_ctrl[4];
    ctrl[3] = h_en_ctrl[6];
    ctrl[4] = h_en_ctrl[5];
    ctrl[5] = h_en_ctrl[7];
  }

  auto tau_wgt = tau->get_wgt(1);
  tau_wgt /= static_cast<double>(electron_pair_list->size());
  tau_wgt /= static_cast<double>(electron_pair_list->size() - 1);
  emp = emp + en2 * tau_wgt;
  if (CVMP2 >= 1) {
    thrust::transform(ctrl.begin(), ctrl.end(), control.begin(), control.begin(), [&](double c, double total) { return total + c * tau_wgt; });
  }
}

template <>
void Device_MP2_Functional<0>::energy(double& emp, std::vector<double>& control, OVPS_Type& ovps, Electron_Pair_List_Type* electron_pair_list, Tau* tau) {
  prep_arrays(ovps, electron_pair_list);
  thrust::fill(scratch_matrix.begin(), scratch_matrix.end(), 0.0);
  mp2_functional_kernal<<<grid_size, block_size>>>(o_direct.data().get(), o_exchange.data().get(), v_direct.data().get(), scratch_matrix.data().get(), vector_size);
  mp2_functional_kernal<<<grid_size, block_size>>>(o_exchange.data().get(), o_direct.data().get(), v_exchange.data().get(), scratch_matrix.data().get(), vector_size);
  cv_energy_helper(0, electron_pair_list->rv_inverse_weight);
  thrust::copy(d_en_ctrl.begin(), d_en_ctrl.end(), h_en_ctrl.begin());
  en2 = h_en_ctrl[0];

  auto tau_wgt = tau->get_wgt(1);
  tau_wgt /= static_cast<double>(electron_pair_list->size());
  tau_wgt /= static_cast<double>(electron_pair_list->size() - 1);
  emp = emp + en2 * tau_wgt;
}
