#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "qc_ovps.cpp"

template <>
void OVPS<thrust::device_vector, thrust::device_allocator>::update(Wavefunction<thrust::device_vector, thrust::device_allocator>& electron_pair_psi1, Wavefunction<thrust::device_vector, thrust::device_allocator>& electron_pair_psi2, Tau* tau) {
  // update green's function trace objects

  auto iocc1 = electron_pair_psi1.iocc1;
  auto iocc2 = electron_pair_psi1.iocc2;
  auto ivir1 = electron_pair_psi1.ivir1;
  auto ivir2 = electron_pair_psi1.ivir2;
  auto lda = electron_pair_psi1.lda;

  hipblasHandle_t handle;
  hipblasCreate(&handle);

  thrust::device_vector<double> psi1 = electron_pair_psi1.psi;
  thrust::device_vector<double> psi2 = electron_pair_psi2.psi;
  thrust::device_vector<double> psi1Tau(psi1.size());
  thrust::device_vector<double> psi2Tau(psi2.size());

  for (auto stop = 0; stop < o_set.size(); stop++) {
    for (auto start = 0; start < o_set[stop].size(); start++) {
      auto t_val = tau->get_exp_tau(stop, start);
      std::transform(t_val.begin(), t_val.end(), t_val.begin(), [](double x){return sqrt(x);});
      thrust::device_vector<double> d_t_val = t_val;

      hipblasDdgmm(handle, HIPBLAS_SIDE_LEFT,
          ivir2 - iocc1, electron_pairs,
          psi1.data().get() + iocc1, lda,
          d_t_val.data().get() + iocc1, 1,
          psi1Tau.data().get() + iocc1, lda);
      hipblasDdgmm(handle, HIPBLAS_SIDE_LEFT,
          ivir2 - iocc1, electron_pairs, 
          psi2.data().get() + iocc1, lda, 
          d_t_val.data().get() + iocc1, 1,
          psi2Tau.data().get() + iocc1, lda);

      o_set[stop][start].update(psi1Tau, iocc1, psi2Tau, iocc1, iocc2 - iocc1, lda);
      v_set[stop][start].update(psi1Tau, ivir1, psi2Tau, ivir1, ivir2 - ivir1, lda);
    }
  }
  hipblasDestroy(handle);
}

void copy_OVPS(OVPS_Host& src, OVPS_Device& dest) {
  for (int i = 0; i < src.o_set.size(); i++) {
    for (int j = 0; j < src.o_set[i].size(); j++) {
      copy_OVPS_Set(src.o_set[i][j], dest.o_set[i][j]);
      copy_OVPS_Set(src.v_set[i][j], dest.v_set[i][j]);
    }
  }
}

void copy_OVPS(OVPS_Device& src, OVPS_Host& dest) {
  for (int i = 0; i < src.o_set.size(); i++) {
    for (int j = 0; j < src.o_set[i].size(); j++) {
      copy_OVPS_Set(src.o_set[i][j], dest.o_set[i][j]);
      copy_OVPS_Set(src.v_set[i][j], dest.v_set[i][j]);
    }
  }
}
