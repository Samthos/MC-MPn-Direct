#include "ovps_set.h"
#include "hipblas.h"

#include "ovps_set.cpp"

template <>
void OVPS_SET_BASE<thrust::device_vector<double>>::update(double *h_psi1Tau, double *h_psi2Tau, size_t inner, size_t lda) {
  double alpha = 1.0;
  double beta = 0.0;

  hipblasHandle_t handle;
  hipblasCreate(&handle);

  double *psi1Tau = nullptr;
  double *psi2Tau = nullptr;
  hipMalloc((void**) &psi1Tau, sizeof(double) * (mc_pair_num)*lda);
  hipMalloc((void**) &psi2Tau, sizeof(double) * (mc_pair_num)*lda);
  hipMemset(psi1Tau, 0, sizeof(double) * mc_pair_num * lda);
  hipMemset(psi2Tau, 0, sizeof(double) * mc_pair_num * lda);
  hipMemcpy(psi1Tau, h_psi1Tau, sizeof(double) * (inner + (mc_pair_num - 1) * lda), hipMemcpyHostToDevice);
  hipMemcpy(psi2Tau, h_psi2Tau, sizeof(double) * (inner + (mc_pair_num - 1) * lda), hipMemcpyHostToDevice);


// printit<<<1,1>>>(psi1Tau, lda);
// hipDeviceSynchronize();
// printf("DEBUGING: inner = %i; lda = %i\n", inner, lda);

  // fill s_12 so upper triangle is zero
  thrust::fill(s_12.begin(), s_12.end(), 0.0);

  // use s_12 as temp storage to produce s_11
  hipblasDsyrk(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_T,
      mc_pair_num, inner, 
      &alpha,
      psi1Tau, lda,
      &beta,
      s_12.data().get(), mc_pair_num);

  // symmetric add s_12 into s_11
  hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_T,
      mc_pair_num, mc_pair_num,
      &alpha,
      s_12.data().get(), mc_pair_num,
      &alpha,
      s_12.data().get(), mc_pair_num,
      s_11.data().get(), mc_pair_num);

  // zero out diagonal
  hipblasDscal(handle, mc_pair_num, &beta, s_11.data().get(), mc_pair_num+1);

  // use s_12 as temp storage to produce s_22
  hipblasDsyrk(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_T,
      mc_pair_num, inner,
      &alpha,
      psi2Tau, lda,
      &beta,
      s_12.data().get(), mc_pair_num);

  // symmetric add s_12 into s_22
  hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_T,
      mc_pair_num, mc_pair_num,
      &alpha,
      s_12.data().get(), mc_pair_num,
      &alpha,
      s_12.data().get(), mc_pair_num,
      s_22.data().get(), mc_pair_num);

  // zero out diagonal
  hipblasDscal(handle, mc_pair_num, &beta, s_22.data().get(), mc_pair_num+1);

  // build s_21
  hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
      mc_pair_num, mc_pair_num, inner,
      &alpha,
      psi1Tau, lda,
      psi2Tau, lda,
      &beta,
      s_21.data().get(), mc_pair_num);

  // zero out diagonal
  hipblasDscal(handle, mc_pair_num, &beta, s_21.data().get(), mc_pair_num+1);

  // set s_12 as transpose of s_21
  hipblasDgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
      mc_pair_num, mc_pair_num,
      &alpha,
      s_21.data().get(), mc_pair_num,
      &beta,
      s_21.data().get(), mc_pair_num,
      s_12.data().get(), mc_pair_num);

  // destroy handle
  hipblasDestroy(handle);
  hipFree(psi1Tau);
  hipFree(psi2Tau);
}

