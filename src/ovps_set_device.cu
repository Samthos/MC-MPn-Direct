#include "hip/hip_runtime.h"

#include "ovps_set.h"
#include "hipblas.h"

#include "ovps_set.cpp"

//__global__ void printit(double* v, int lda) {
//  for (int i = 0; i < lda; i++) {
//    printf("DEBUGING: psi %2i %12.4f %12.4f\n", i, v[i], v[i + lda]);
//  }
//  printf("DEBUGING\n");
//}

template <>
void OVPS_SET_BASE<thrust::device_vector<double>>::update(double *h_psi1Tau, double *h_psi2Tau, size_t inner, size_t lda) {
  double alpha = 1.0;
  double beta = 0.0;

  hipblasHandle_t handle;
  hipblasCreate(&handle);

  double *psi1Tau = nullptr;
  double *psi2Tau = nullptr;
  hipMalloc((void**) &psi1Tau, sizeof(double) * (mc_pair_num)*lda);
  hipMalloc((void**) &psi2Tau, sizeof(double) * (mc_pair_num)*lda);
  hipMemset(psi1Tau, 0, sizeof(double) * mc_pair_num * lda);
  hipMemset(psi2Tau, 0, sizeof(double) * mc_pair_num * lda);
  hipMemcpy(psi1Tau, h_psi1Tau, sizeof(double) * (inner + (mc_pair_num - 1) * lda), hipMemcpyHostToDevice);
  hipMemcpy(psi2Tau, h_psi2Tau, sizeof(double) * (inner + (mc_pair_num - 1) * lda), hipMemcpyHostToDevice);


// printit<<<1,1>>>(psi1Tau, lda);
// hipDeviceSynchronize();
// printf("DEBUGING: inner = %i; lda = %i\n", inner, lda);

  // fill s_12 so upper triangle is zero
  thrust::fill(s_12.begin(), s_12.end(), 0.0);

  // use s_12 as temp storage to produce s_11
  hipblasDsyrk(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_T,
      mc_pair_num, inner, 
      &alpha,
      psi1Tau, lda,
      &beta,
      s_12.data().get(), mc_pair_num);

  // symmetric add s_12 into s_11
  hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_T,
      mc_pair_num, mc_pair_num,
      &alpha,
      s_12.data().get(), mc_pair_num,
      &alpha,
      s_12.data().get(), mc_pair_num,
      s_11.data().get(), mc_pair_num);

  // zero out diagonal
  hipblasDscal(handle, mc_pair_num, &beta, s_11.data().get(), mc_pair_num+1);

  // use s_12 as temp storage to produce s_22
  hipblasDsyrk(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_T,
      mc_pair_num, inner,
      &alpha,
      psi2Tau, lda,
      &beta,
      s_12.data().get(), mc_pair_num);

  // symmetric add s_12 into s_22
  hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_T,
      mc_pair_num, mc_pair_num,
      &alpha,
      s_12.data().get(), mc_pair_num,
      &alpha,
      s_12.data().get(), mc_pair_num,
      s_22.data().get(), mc_pair_num);

  // zero out diagonal
  hipblasDscal(handle, mc_pair_num, &beta, s_22.data().get(), mc_pair_num+1);

  // build s_21
  hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
      mc_pair_num, mc_pair_num, inner,
      &alpha,
      psi1Tau, lda,
      psi2Tau, lda,
      &beta,
      s_21.data().get(), mc_pair_num);

  // zero out diagonal
  hipblasDscal(handle, mc_pair_num, &beta, s_21.data().get(), mc_pair_num+1);

  // set s_12 as transpose of s_21
  hipblasDgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
      mc_pair_num, mc_pair_num,
      &alpha,
      s_21.data().get(), mc_pair_num,
      &beta,
      s_21.data().get(), mc_pair_num,
      s_12.data().get(), mc_pair_num);

  // destroy handle
  hipblasDestroy(handle);
  hipFree(psi1Tau);
  hipFree(psi2Tau);
}

void copy_OVPS_HOST_TO_DEVICE(OVPS_SET& src, OVPS_SET_DEVICE& dest) {
  thrust::copy(src.s_11.begin(), src.s_11.end(), dest.s_11.begin());
  thrust::copy(src.s_12.begin(), src.s_12.end(), dest.s_12.begin());
  thrust::copy(src.s_21.begin(), src.s_21.end(), dest.s_21.begin());
  thrust::copy(src.s_22.begin(), src.s_22.end(), dest.s_22.begin());
}

void copy_OVPS_DEVICE_TO_HOST(OVPS_SET_DEVICE& src, OVPS_SET& dest) {
  thrust::copy(src.s_11.begin(), src.s_11.end(), dest.s_11.begin());
  thrust::copy(src.s_12.begin(), src.s_12.end(), dest.s_12.begin());
  thrust::copy(src.s_21.begin(), src.s_21.end(), dest.s_21.begin());
  thrust::copy(src.s_22.begin(), src.s_22.end(), dest.s_22.begin());
}
