#include "hipblas.h"
#include "ovps_set.cpp"

template <>
void OVPS_Set_Device::update(vector_double& psi1Tau, int psi1_offset, vector_double& psi2Tau, int psi2_offset, size_t inner, size_t lda) {
  double alpha = 1.0;
  double beta = 0.0;

  hipblasHandle_t handle;
  hipblasCreate(&handle);

  // fill s_12 so upper triangle is zero
  thrust::fill(s_12.begin(), s_12.end(), 0.0);

  // use s_12 as temp storage to produce s_11
  hipblasDsyrk(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_T,
      mc_pair_num, inner, 
      &alpha,
      psi1Tau.data().get() + psi1_offset, lda,
      &beta,
      s_12.data().get(), mc_pair_num);

  // symmetric add s_12 into s_11
  hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_T,
      mc_pair_num, mc_pair_num,
      &alpha,
      s_12.data().get(), mc_pair_num,
      &alpha,
      s_12.data().get(), mc_pair_num,
      s_11.data().get(), mc_pair_num);

  // zero out diagonal
  hipblasDscal(handle, mc_pair_num, &beta, s_11.data().get(), mc_pair_num+1);

  // use s_12 as temp storage to produce s_22
  hipblasDsyrk(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_T,
      mc_pair_num, inner,
      &alpha,
      psi2Tau.data().get() + psi2_offset, lda,
      &beta,
      s_12.data().get(), mc_pair_num);

  // symmetric add s_12 into s_22
  hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_T,
      mc_pair_num, mc_pair_num,
      &alpha,
      s_12.data().get(), mc_pair_num,
      &alpha,
      s_12.data().get(), mc_pair_num,
      s_22.data().get(), mc_pair_num);

  // zero out diagonal
  hipblasDscal(handle, mc_pair_num, &beta, s_22.data().get(), mc_pair_num+1);

  // build s_21
  hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
      mc_pair_num, mc_pair_num, inner,
      &alpha,
      psi1Tau.data().get() + psi1_offset, lda,
      psi2Tau.data().get() + psi2_offset, lda,
      &beta,
      s_21.data().get(), mc_pair_num);

  // zero out diagonal
  hipblasDscal(handle, mc_pair_num, &beta, s_21.data().get(), mc_pair_num+1);

  // set s_12 as transpose of s_21
  hipblasDgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
      mc_pair_num, mc_pair_num,
      &alpha,
      s_21.data().get(), mc_pair_num,
      &beta,
      s_21.data().get(), mc_pair_num,
      s_12.data().get(), mc_pair_num);

  // destroy handle
  hipblasDestroy(handle);
}

void copy_OVPS_Set(OVPS_Set_Host& src, OVPS_Set_Device& dest) {
  thrust::copy(src.s_11.begin(), src.s_11.end(), dest.s_11.begin());
  thrust::copy(src.s_12.begin(), src.s_12.end(), dest.s_12.begin());
  thrust::copy(src.s_21.begin(), src.s_21.end(), dest.s_21.begin());
  thrust::copy(src.s_22.begin(), src.s_22.end(), dest.s_22.begin());
}

void copy_OVPS_Set(OVPS_Set_Device& src, OVPS_Set_Host& dest) {
  thrust::copy(src.s_11.begin(), src.s_11.end(), dest.s_11.begin());
  thrust::copy(src.s_12.begin(), src.s_12.end(), dest.s_12.begin());
  thrust::copy(src.s_21.begin(), src.s_21.end(), dest.s_21.begin());
  thrust::copy(src.s_22.begin(), src.s_22.end(), dest.s_22.begin());
}
